#include "PathTracer.cuh"
#include <cuda_gl_interop.h>
#include <iostream>

#include "kernels.cuh"

PathTracer::PathTracer(const GLuint glTexture, const int pixelWidth, const int pixelHeight) : _width(pixelWidth), _height(pixelHeight) {
	hipError_t cudaStatus;

	uint cudaDeviceCount;
	int cudaDevices[4];
	cudaStatus = hipGLGetDevices(&cudaDeviceCount, cudaDevices, 4, hipGLDeviceList::hipGLDeviceListAll);
	_CheckCudaError(cudaStatus, "hipGLGetDevices");

	std::cout << "Found " << cudaDeviceCount << " CUDA-capable devices, using first device (device " << cudaDevices[0] << ")" << std::endl;

	cudaStatus = hipSetDevice(cudaDevices[0]);
	_CheckCudaError(cudaStatus, "hipSetDevice");
	
	cudaStatus = hipGraphicsGLRegisterImage(&this->_cudaTexture, glTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsSurfaceLoadStore);
	_CheckCudaError(cudaStatus, "hipGraphicsGLRegisterImage");
}

PathTracer::~PathTracer() {
	std::cout << "Resetting CUDA device" << std::endl;
	hipError_t cudaStatus = hipDeviceReset();
	_CheckCudaError(cudaStatus, "hipDeviceReset");
}

void PathTracer::Update() {

}

void PathTracer::Draw() {
	std::cout << "Drawing CudaTest path tracer" << std::endl;
	hipError_t cudaStatus;
	cudaStatus = hipGraphicsMapResources(1, &this->_cudaTexture);
	_CheckCudaError(cudaStatus, "hipGraphicsMapResources");

	hipArray_t abcdef;
	cudaStatus = hipGraphicsSubResourceGetMappedArray(&abcdef, this->_cudaTexture, 0, 0);
	_CheckCudaError(cudaStatus, "hipGraphicsResourceGetMappedPointer");

	hipSurfaceObject_t surface;
	hipResourceDesc resourceDesc;
	resourceDesc.resType = hipResourceType::hipResourceTypeArray;
	resourceDesc.res.array.array = abcdef;
	cudaStatus = hipCreateSurfaceObject(&surface, &resourceDesc);
	_CheckCudaError(cudaStatus, "hipCreateSurfaceObject");

	DrawToTexture<<<16, 16>>>(surface, this->_width, this->_height);

	cudaStatus = hipDestroySurfaceObject(surface);
	_CheckCudaError(cudaStatus, "hipDestroySurfaceObject");

	cudaStatus = hipGraphicsUnmapResources(1, &this->_cudaTexture);
	_CheckCudaError(cudaStatus, "hipGraphicsUnmapResources");
}

void PathTracer::Resize(const int pixelWidth, const int pixelHeight) {

}

void PathTracer::_CheckCudaError(const hipError_t cudaStatus, const char* functionName) {
	if (cudaStatus != hipError_t::hipSuccess) {
		std::cout << "Failed to execute '" << functionName << "' (error " << cudaStatus << ")" << std::endl;
		throw std::exception();
	}
}

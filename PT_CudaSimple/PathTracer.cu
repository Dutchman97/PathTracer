#include "PathTracer.cuh"
#include <cuda_gl_interop.h>
#include <iostream>

PathTracer::PathTracer(const GLuint glTexture, const int pixelWidth, const int pixelHeight) : _width(pixelWidth), _height(pixelHeight) {
	hipError_t cudaStatus;

	uint cudaDeviceCount;
	int cudaDevices[4];
	cudaStatus = hipGLGetDevices(&cudaDeviceCount, cudaDevices, 4, hipGLDeviceList::hipGLDeviceListAll);
	_CheckCudaError(cudaStatus, "hipGLGetDevices");

	std::cout << "Found " << cudaDeviceCount << " CUDA-capable devices, using first device (device " << cudaDevices[0] << ")" << std::endl;

	cudaStatus = hipSetDevice(cudaDevices[0]);
	_CheckCudaError(cudaStatus, "hipSetDevice");
	
	cudaStatus = hipGraphicsGLRegisterImage(&this->_cudaTexture, glTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsSurfaceLoadStore);
	_CheckCudaError(cudaStatus, "hipGraphicsGLRegisterImage");
}

PathTracer::~PathTracer() {
	std::cout << "Resetting CUDA device" << std::endl;
	hipError_t cudaStatus = hipDeviceReset();
	_CheckCudaError(cudaStatus, "hipDeviceReset");
}

void PathTracer::Update() {

}

void PathTracer::Draw() {

}

void PathTracer::Resize(const int pixelWidth, const int pixelHeight) {

}

void PathTracer::_CheckCudaError(const hipError_t cudaStatus, const char* functionName) {
	if (cudaStatus != hipError_t::hipSuccess) {
		std::cout << "Failed to execute '" << functionName << "' (error " << cudaStatus << ")" << std::endl;
		throw std::exception();
	}
}

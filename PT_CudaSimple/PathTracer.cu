#include "hip/hip_runtime.h"
#include "PathTracer.cuh"
#include <cuda_gl_interop.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#include "kernels.cuh"

constexpr int TEST_WIDTH = 8, TEST_HEIGHT = 8;

PathTracer::PathTracer(const GLuint glTexture, const int pixelWidth, const int pixelHeight) : _width(pixelWidth), _height(pixelHeight), _glTexture(glTexture) {
	hipError_t cudaStatus;
	hiprandStatus_t hiprandStatus;

	uint cudaDeviceCount;
	int cudaDevices[4];
	cudaStatus = hipGLGetDevices(&cudaDeviceCount, cudaDevices, 4, hipGLDeviceList::hipGLDeviceListAll);
	_CheckCudaError(cudaStatus, "hipGLGetDevices");

	std::cout << "Found " << cudaDeviceCount << " CUDA-capable devices linked to the current OpenGL context, using first device (device " << cudaDevices[0] << ")" << std::endl;

	this->_PrintDeviceInfo(cudaDevices[0]);

	cudaStatus = hipSetDevice(cudaDevices[0]);
	_CheckCudaError(cudaStatus, "hipSetDevice");

	cudaStatus = hipMallocPitch(&this->_drawingVariables.devicePtrs.rays, &this->_drawingVariables.devicePtrs.rayArrayPitch, pixelWidth * sizeof(Ray), pixelHeight);
	_CheckCudaError(cudaStatus, "hipMallocPitch");

	hiprandStateXORWOW_t* rngStates;
	size_t rngStatesPitch;
	cudaStatus = hipMallocPitch(&rngStates, &rngStatesPitch, pixelWidth * sizeof(hiprandStateXORWOW_t), pixelHeight);
	_CheckCudaError(cudaStatus, "hipMallocPitch");

	dim3 threadsPerBlock(TEST_WIDTH, TEST_HEIGHT);
	InitializeRng<<<1, threadsPerBlock>>>(rngStates, rngStatesPitch);

	float* rngValuesDevice;
	cudaStatus = hipMalloc(&rngValuesDevice, TEST_WIDTH * TEST_HEIGHT * sizeof(float));
	_CheckCudaError(cudaStatus, "hipMalloc");

	TestRng<<<1, threadsPerBlock>>>(rngStates, rngStatesPitch, rngValuesDevice);

	cudaStatus = hipDeviceSynchronize();
	_CheckCudaError(cudaStatus, "hipDeviceSynchronize");

	float* rngValues = (float*)calloc(TEST_WIDTH * TEST_HEIGHT, sizeof(float));
	cudaStatus = hipMemcpy(rngValues, rngValuesDevice, sizeof(float) * TEST_WIDTH * TEST_HEIGHT, hipMemcpyKind::hipMemcpyDeviceToHost);
	_CheckCudaError(cudaStatus, "hipMemcpy");

	for (int i = 0; i < TEST_WIDTH * TEST_HEIGHT; i++) {
		std::cout << rngValues[i] << std::endl;
	}

	cudaStatus = hipFree(rngValuesDevice);
	_CheckCudaError(cudaStatus, "hipFree");
	free(rngValues);

	//hiprandGenerator_t curandGenerator;
	//hiprandStatus = hiprandCreateGenerator(&curandGenerator, hiprandRngType_t::HIPRAND_RNG_PSEUDO_XORWOW);
	//_CheckCurandError(hiprandStatus, "hiprandCreateGenerator");

	//hiprandStatus = hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 1337);
	//_CheckCurandError(hiprandStatus, "hiprandSetPseudoRandomGeneratorSeed");


	//hiprandStatus = hiprandDestroyGenerator(curandGenerator);
	//_CheckCurandError(hiprandStatus, "hiprandDestroyGenerator");
}

PathTracer::~PathTracer() {
	hipError_t cudaStatus;
	std::cout << "Terminating CUDA path tracer" << std::endl;

	if (this->_drawingVariables.drawState == DrawState::Drawing) {
		this->FinalizeDrawing();
	}

	cudaStatus = hipFree(this->_drawingVariables.devicePtrs.rays);
	_CheckCudaError(cudaStatus, "hipFree");

	cudaStatus = hipDeviceReset();
	_CheckCudaError(cudaStatus, "hipDeviceReset");
}

void PathTracer::Update() {

}

void PathTracer::BeginDrawing() {
	hipError_t cudaStatus;
	this->_drawingVariables.drawState = DrawState::Drawing;

	this->_MapTexture(
		this->_glTexture,
		&this->_drawingVariables.cudaTextureResource,
		&this->_drawingVariables.cudaSurface
	);

	dim3 threadsPerBlock(TEST_WIDTH, TEST_HEIGHT);
	//DrawToTexture<<<1, threadsPerBlock>>>(this->_drawingVariables.cudaSurface);
	InitializeRays<<<1, threadsPerBlock>>>(
		this->_drawingVariables.devicePtrs.rays, this->_drawingVariables.devicePtrs.rayArrayPitch,
		TEST_WIDTH, TEST_HEIGHT,
		//this->_width, this->_height,
		make_float4(0.0f, 0.0f, 0.0f, 0.0f),
		make_float4(-1.0f, 1.0f, 1.0f, 0.0f),
		make_float4(1.0f, 1.0f, 1.0f, 0.0f),
		make_float4(-1.0f, -1.0f, 1.0f, 0.0f)
	);

	cudaStatus = hipGetLastError();
	_CheckCudaError(cudaStatus, "hipGetLastError");
}

void PathTracer::FinalizeDrawing() {
	if (this->_drawingVariables.drawState != DrawState::Drawing) {
		return;
	}
	this->_drawingVariables.drawState = DrawState::Idle;

	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	_CheckCudaError(cudaStatus, "hipDeviceSynchronize");

	this->_UnmapTexture(
		&this->_drawingVariables.cudaTextureResource,
		&this->_drawingVariables.cudaSurface
	);
}

void PathTracer::Resize(const int pixelWidth, const int pixelHeight) {

}

inline void PathTracer::_CheckCudaError(const hipError_t cudaStatus, const char* functionName) {
	if (cudaStatus != hipError_t::hipSuccess) {
		std::cout << "Failed to execute '" << functionName << "' (" << hipGetErrorName(cudaStatus) << ")" << std::endl <<
			"\t" << hipGetErrorString(cudaStatus) << std::endl;
		throw std::exception();
	}
}

inline void PathTracer::_CheckCurandError(const hiprandStatus_t hiprandStatus, const char* functionName) {
	if (hiprandStatus != hiprandStatus_t::HIPRAND_STATUS_SUCCESS) {
		std::string errorName;
		switch (hiprandStatus) {
		case HIPRAND_STATUS_VERSION_MISMATCH: ///< Header file and linked library version do not match
			errorName = "HIPRAND_STATUS_VERSION_MISMATCH";
			break;
		case HIPRAND_STATUS_NOT_INITIALIZED: ///< Generator not initialized
			errorName = "HIPRAND_STATUS_NOT_INITIALIZED";
			break;
		case HIPRAND_STATUS_ALLOCATION_FAILED: ///< Memory allocation failed
			errorName = "HIPRAND_STATUS_ALLOCATION_FAILED";
			break;
		case HIPRAND_STATUS_TYPE_ERROR: ///< Generator is wrong type
			errorName = "HIPRAND_STATUS_TYPE_ERROR";
			break;
		case HIPRAND_STATUS_OUT_OF_RANGE: ///< Argument out of range
			errorName = "HIPRAND_STATUS_OUT_OF_RANGE";
			break;
		case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE: ///< Length requested is not a multple of dimension
			errorName = "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
			break;
		case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED: ///< GPU does not have double precision required by MRG32k3a
			errorName = "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
			break;
		case HIPRAND_STATUS_LAUNCH_FAILURE: ///< Kernel launch failure
			errorName = "HIPRAND_STATUS_LAUNCH_FAILURE";
			break;
		case HIPRAND_STATUS_PREEXISTING_FAILURE: ///< Preexisting failure on library entry
			errorName = "HIPRAND_STATUS_PREEXISTING_FAILURE";
			break;
		case HIPRAND_STATUS_INITIALIZATION_FAILED: ///< Initialization of CUDA failed
			errorName = "HIPRAND_STATUS_INITIALIZATION_FAILED";
			break;
		case HIPRAND_STATUS_ARCH_MISMATCH: ///< Architecture mismatch, GPU does not support requested feature
			errorName = "HIPRAND_STATUS_ARCH_MISMATCH";
			break;
		case HIPRAND_STATUS_INTERNAL_ERROR: ///< Internal library error
			errorName = "HIPRAND_STATUS_INTERNAL_ERROR";
			break;
		default:
			errorName = "Unknown error";
			break;
		}
		std::cout << "Failed to execute '" << functionName << "' (" << errorName.c_str() << ")" << std::endl;
		throw std::exception();
	}
}

void PathTracer::_MapTexture(const GLuint glTexture, hipGraphicsResource_t* cudaResourcePtr, hipSurfaceObject_t* cudaSurfacePtr) const {
	hipError_t cudaStatus;
	cudaStatus = hipGraphicsGLRegisterImage(cudaResourcePtr, glTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsSurfaceLoadStore);
	_CheckCudaError(cudaStatus, "hipGraphicsGLRegisterImage");

	cudaStatus = hipGraphicsMapResources(1, cudaResourcePtr);
	_CheckCudaError(cudaStatus, "hipGraphicsMapResources");

	hipArray_t abcdef;
	cudaStatus = hipGraphicsSubResourceGetMappedArray(&abcdef, *cudaResourcePtr, 0, 0);
	_CheckCudaError(cudaStatus, "hipGraphicsResourceGetMappedPointer");

	hipResourceDesc resourceDesc = hipResourceDesc();
	resourceDesc.resType = hipResourceType::hipResourceTypeArray;
	resourceDesc.res.array.array = abcdef;
	cudaStatus = hipCreateSurfaceObject(cudaSurfacePtr, &resourceDesc);
	_CheckCudaError(cudaStatus, "hipCreateSurfaceObject");
}

void PathTracer::_UnmapTexture(hipGraphicsResource_t* cudaResourcePtr, hipSurfaceObject_t* cudaSurfacePtr) const {
	hipError_t cudaStatus;
	cudaStatus = hipDestroySurfaceObject(*cudaSurfacePtr);
	_CheckCudaError(cudaStatus, "hipDestroySurfaceObject");

	cudaStatus = hipGraphicsUnmapResources(1, cudaResourcePtr);
	_CheckCudaError(cudaStatus, "hipGraphicsUnmapResources");

	cudaStatus = hipGraphicsUnregisterResource(*cudaResourcePtr);
	_CheckCudaError(cudaStatus, "hipGraphicsUnregisterResource");
}

void PathTracer::_PrintDeviceInfo(const int device) const {
	hipError_t cudaStatus;
	hipDeviceProp_t properties;
	cudaStatus = hipGetDeviceProperties(&properties, device);
	_CheckCudaError(cudaStatus, "hipGetDeviceProperties");

	std::printf("Using '%s'\n", properties.name);
	std::printf("\tCompute capability:      %i.%i\n", properties.major, properties.minor);
	std::printf("\tMultiprocessors:         %i\n", properties.multiProcessorCount);
	std::printf("\tWarp size:               %i\n", properties.warpSize);
	std::printf("\tConcurrent engine count: %i\n", properties.asyncEngineCount);
}

#include "PathTracer.cuh"
#include <cuda_gl_interop.h>
#include <iostream>

#include "kernels.cuh"

PathTracer::PathTracer(const GLuint glTexture, const int pixelWidth, const int pixelHeight) : _width(pixelWidth), _height(pixelHeight), _glTexture(glTexture) {
	hipError_t cudaStatus;

	uint cudaDeviceCount;
	int cudaDevices[4];
	cudaStatus = hipGLGetDevices(&cudaDeviceCount, cudaDevices, 4, hipGLDeviceList::hipGLDeviceListAll);
	_CheckCudaError(cudaStatus, "hipGLGetDevices");

	std::cout << "Found " << cudaDeviceCount << " CUDA-capable devices linked to the current OpenGL context, using first device (device " << cudaDevices[0] << ")" << std::endl;

	this->_PrintDeviceInfo(cudaDevices[0]);

	cudaStatus = hipSetDevice(cudaDevices[0]);
	_CheckCudaError(cudaStatus, "hipSetDevice");
}

PathTracer::~PathTracer() {
	std::cout << "Terminating CUDA path tracer" << std::endl;

	if (this->_drawingVariables.drawState == DrawState::Drawing) {
		this->FinalizeDrawing();
	}

	hipError_t cudaStatus = hipDeviceReset();
	_CheckCudaError(cudaStatus, "hipDeviceReset");
}

void PathTracer::Update() {

}

void PathTracer::BeginDrawing() {
	hipError_t cudaStatus;
	this->_drawingVariables = DrawingVariables();
	this->_drawingVariables.drawState = DrawState::Drawing;

	this->_MapTexture(
		this->_glTexture,
		&this->_drawingVariables.cudaTextureResource,
		&this->_drawingVariables.cudaSurface
	);

	dim3 threadsPerBlock(16, 16);
	DrawToTexture<<<1, threadsPerBlock>>>(this->_drawingVariables.cudaSurface);

	cudaStatus = hipGetLastError();
	_CheckCudaError(cudaStatus, "hipGetLastError");
}

void PathTracer::FinalizeDrawing() {
	if (this->_drawingVariables.drawState != DrawState::Drawing) {
		return;
	}
	this->_drawingVariables.drawState = DrawState::Idle;

	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	_CheckCudaError(cudaStatus, "hipDeviceSynchronize");

	this->_UnmapTexture(
		&this->_drawingVariables.cudaTextureResource,
		&this->_drawingVariables.cudaSurface
	);
}

void PathTracer::Resize(const int pixelWidth, const int pixelHeight) {

}

inline void PathTracer::_CheckCudaError(const hipError_t cudaStatus, const char* functionName) {
	if (cudaStatus != hipError_t::hipSuccess) {
		std::cout << "Failed to execute '" << functionName << "' (" << hipGetErrorName(cudaStatus) << ")" << std::endl <<
			"\t" << hipGetErrorString(cudaStatus) << std::endl;
		throw std::exception();
	}
}

void PathTracer::_MapTexture(const GLuint glTexture, hipGraphicsResource_t* cudaResourcePtr, hipSurfaceObject_t* cudaSurfacePtr) const {
	hipError_t cudaStatus;
	cudaStatus = hipGraphicsGLRegisterImage(cudaResourcePtr, glTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsSurfaceLoadStore);
	_CheckCudaError(cudaStatus, "hipGraphicsGLRegisterImage");

	cudaStatus = hipGraphicsMapResources(1, cudaResourcePtr);
	_CheckCudaError(cudaStatus, "hipGraphicsMapResources");

	hipArray_t abcdef;
	cudaStatus = hipGraphicsSubResourceGetMappedArray(&abcdef, *cudaResourcePtr, 0, 0);
	_CheckCudaError(cudaStatus, "hipGraphicsResourceGetMappedPointer");

	hipResourceDesc resourceDesc = hipResourceDesc();
	resourceDesc.resType = hipResourceType::hipResourceTypeArray;
	resourceDesc.res.array.array = abcdef;
	cudaStatus = hipCreateSurfaceObject(cudaSurfacePtr, &resourceDesc);
	_CheckCudaError(cudaStatus, "hipCreateSurfaceObject");
}

void PathTracer::_UnmapTexture(hipGraphicsResource_t* cudaResourcePtr, hipSurfaceObject_t* cudaSurfacePtr) const {
	hipError_t cudaStatus;
	cudaStatus = hipDestroySurfaceObject(*cudaSurfacePtr);
	_CheckCudaError(cudaStatus, "hipDestroySurfaceObject");

	cudaStatus = hipGraphicsUnmapResources(1, cudaResourcePtr);
	_CheckCudaError(cudaStatus, "hipGraphicsUnmapResources");

	cudaStatus = hipGraphicsUnregisterResource(*cudaResourcePtr);
	_CheckCudaError(cudaStatus, "hipGraphicsUnregisterResource");
}

void PathTracer::_PrintDeviceInfo(const int device) const {
	hipError_t cudaStatus;
	hipDeviceProp_t properties;
	cudaStatus = hipGetDeviceProperties(&properties, device);
	_CheckCudaError(cudaStatus, "hipGetDeviceProperties");

	std::printf("Using '%s'\n", properties.name);
	std::printf("\tCompute capability:      %i.%i\n", properties.major, properties.minor);
	std::printf("\tMultiprocessors:         %i\n", properties.multiProcessorCount);
	std::printf("\tWarp size:               %i\n", properties.warpSize);
	std::printf("\tConcurrent engine count: %i\n", properties.asyncEngineCount);
}

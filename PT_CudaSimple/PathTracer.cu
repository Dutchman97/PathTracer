#include "PathTracer.cuh"
#include <cuda_gl_interop.h>
#include <iostream>

#include "kernels.cuh"

PathTracer::PathTracer(const GLuint glTexture, const int pixelWidth, const int pixelHeight) : _width(pixelWidth), _height(pixelHeight), _glTexture(glTexture) {
	hipError_t cudaStatus;

	uint cudaDeviceCount;
	int cudaDevices[4];
	cudaStatus = hipGLGetDevices(&cudaDeviceCount, cudaDevices, 4, hipGLDeviceList::hipGLDeviceListAll);
	_CheckCudaError(cudaStatus, "hipGLGetDevices");

	std::cout << "Found " << cudaDeviceCount << " CUDA-capable devices, using first device (device " << cudaDevices[0] << ")" << std::endl;

	cudaStatus = hipSetDevice(cudaDevices[0]);
	_CheckCudaError(cudaStatus, "hipSetDevice");
}

PathTracer::~PathTracer() {
	std::cout << "Resetting CUDA device" << std::endl;
	hipError_t cudaStatus = hipDeviceReset();
	_CheckCudaError(cudaStatus, "hipDeviceReset");
}

void PathTracer::Update() {

}

void PathTracer::Draw() {
	hipError_t cudaStatus;
	cudaStatus = hipGraphicsGLRegisterImage(&this->_cudaTexture, this->_glTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsSurfaceLoadStore);
	_CheckCudaError(cudaStatus, "hipGraphicsGLRegisterImage");

	cudaStatus = hipGraphicsMapResources(1, &this->_cudaTexture);
	_CheckCudaError(cudaStatus, "hipGraphicsMapResources");

	hipArray_t abcdef;
	cudaStatus = hipGraphicsSubResourceGetMappedArray(&abcdef, this->_cudaTexture, 0, 0);
	_CheckCudaError(cudaStatus, "hipGraphicsResourceGetMappedPointer");

	hipSurfaceObject_t surface;
	hipResourceDesc resourceDesc = hipResourceDesc();
	resourceDesc.resType = hipResourceType::hipResourceTypeArray;
	resourceDesc.res.array.array = abcdef;
	cudaStatus = hipCreateSurfaceObject(&surface, &resourceDesc);
	_CheckCudaError(cudaStatus, "hipCreateSurfaceObject");

	dim3 threadsPerBlock(16, 16);
	DrawToTexture<<<1, threadsPerBlock>>>(surface);

	cudaStatus = hipGetLastError();
	_CheckCudaError(cudaStatus, "hipGetLastError");

	cudaStatus = hipDeviceSynchronize();
	_CheckCudaError(cudaStatus, "hipDeviceSynchronize");

	cudaStatus = hipDestroySurfaceObject(surface);
	_CheckCudaError(cudaStatus, "hipDestroySurfaceObject");

	cudaStatus = hipGraphicsUnmapResources(1, &this->_cudaTexture);
	_CheckCudaError(cudaStatus, "hipGraphicsUnmapResources");

	cudaStatus = hipGraphicsUnregisterResource(this->_cudaTexture);
	_CheckCudaError(cudaStatus, "hipGraphicsUnregisterResource");
}

void PathTracer::Resize(const int pixelWidth, const int pixelHeight) {

}

inline void PathTracer::_CheckCudaError(const hipError_t cudaStatus, const char* functionName) {
	if (cudaStatus != hipError_t::hipSuccess) {
		std::cout << "Failed to execute '" << functionName << "' (error " << cudaStatus << ")" << std::endl;
		throw std::exception();
	}
}

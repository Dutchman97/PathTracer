#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"

__global__ void DrawToTexture(hipSurfaceObject_t texture, int screenWidth, int screenHeight, Intersection* intersections, uint frameNumber, float4* frameBuffer) {
	uint i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= screenWidth * screenHeight) return;

	uint x = i % screenWidth;
	uint y = i / screenWidth;

	//float t = intersections[i].t;
	//float red = (t > EPSILON && t < FLT_MAX) ? 1.0f : 0.2f;
	//float4 color = make_float4(red, 0.2f, 0.2f, 1.0f);

	float4 color = frameBuffer[i];

	// IMPORTANT: Surface functions use bytes for addressing memory; x-coordinate is in bytes.
	// Y-coordinate does not need to be multiplied as the byte offset of the corresponding y-coordinate is internally calculated.
	float4 previousColor;
	surf2Dread(&previousColor, texture, x * sizeof(float4), y);
	surf2Dwrite(color / (frameNumber + 1) + previousColor * frameNumber / (frameNumber + 1), texture, x * sizeof(float4), y);
}

__global__ void InitializeRng(hiprandStateXORWOW_t* rngStates, int count) {
	uint i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= count) return;

	hiprand_init(1337 + i, 0, 0, &rngStates[i]);
}

__global__ void InitializeRays(Ray* rays, hiprandStateXORWOW_t* rngStates, int screenWidth, int screenHeight, float4 origin, float4 topLeft, float4 bottomLeft, float4 bottomRight, Intersection* intersections, float4* frameBuffer) {
	uint i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= screenWidth * screenHeight) return;

	uint x = i % screenWidth;
	uint y = i / screenWidth;

	float xScreen = ((float)x + hiprand_uniform(&rngStates[i])) / screenWidth;
	float yScreen = ((float)y + hiprand_uniform(&rngStates[i])) / screenHeight;

	Ray* rayPtr = &rays[i];
	rayPtr->origin = origin;
	rayPtr->direction = normalize(bottomLeft + (bottomRight - bottomLeft) * xScreen + (topLeft - bottomLeft) * yScreen);

	intersections[i] = NO_INTERSECTION;

	frameBuffer[i] = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
}

// Uses the intersection algorithm by Möller and Trumbore.
// https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
__device__ Intersection RayIntersectsTriangle(Ray* rayPtr, Triangle* trianglePtr, Vertex* vertices) {
	int idx0 = trianglePtr->vertexIdx0;
	int idx1 = trianglePtr->vertexIdx1;
	int idx2 = trianglePtr->vertexIdx2;

	float4 v0 = vertices[idx0].position;
	float4 v1 = vertices[idx1].position;
	float4 v2 = vertices[idx2].position;

	float4 edge0 = v1 - v0;
	float4 edge1 = v2 - v0;

	float4 h = cross(rayPtr->direction, edge1);
	float determinant = dot(edge0, h);

#ifdef CULLING_ENABLED
	if (determinant < EPSILON) return NO_INTERSECTION;
#else
	if (determinant > -EPSILON || determinant < EPSILON) return 0.0f;
#endif

	float f = 1.0f / determinant;
	float4 s = rayPtr->origin - v0;

	float u = f * dot(s, h);
	if (u < 0.0f || u > 1.0f) return NO_INTERSECTION;

	float4 q = cross(s, edge0);
	float v = f * dot(rayPtr->direction, q);
	if (v < 0.0f || u + v > 1.0f) return NO_INTERSECTION;

	float t = f * dot(edge1, q);
	return Intersection { t, trianglePtr->materialIdx, cross(edge0, edge1) };
}

__global__ void TraverseScene(Ray* rays, int rayCount, Triangle* triangles, int triangleCount, Vertex* vertices, Intersection* intersections) {
	uint rayIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (rayIdx >= rayCount || rays[rayIdx].direction == ZERO_VECTOR) return;

	for (int triangleIdx = 0; triangleIdx < triangleCount; triangleIdx++) {
		Intersection intersection = RayIntersectsTriangle(&rays[rayIdx], &triangles[triangleIdx], vertices);
		if (intersection.t > EPSILON && intersection.t < intersections[rayIdx].t) {
			intersections[rayIdx] = intersection;
		}
	}
}

__device__ float4 GetDiffuseReflection(float4 normal, hiprandStateXORWOW_t* rngStatePtr) {
	// Generate a random vector.
	// We want to make sure this random vector is not longer than 1, because we normalize this vector,
	// the resulting vector may skew towards the corners of the unit cube because of how we obtain this random vector.
	// Also ensure this loop does execture infinitely.
	// This still causes a slight skew towards the corners, so a better way to obtain a random unit vector is necessary.
	float4 result;
	uint loopCounter = 0;
	do {
		result = make_float4(hiprand_uniform(rngStatePtr), hiprand_uniform(rngStatePtr), hiprand_uniform(rngStatePtr), 0.0f);
		result = result * 2.0f - 1.0f;

		loopCounter++;
	} while (lengthSquared(result) > 1.0f && loopCounter <= 10);

	// Normalize the result and ensure it's pointing in the normal vector's hemisphere half.
	result = normalize(result);
	return dot(result, normal) > 0.0f ? result : -result;
}

__global__ void Intersect(Ray* rays, int rayCount, Intersection* intersections, Material* materials, hiprandStateXORWOW_t* rngStates, float4* frameBuffer) {
	uint rayIdx = threadIdx.x + blockIdx.x * blockDim.x;

	if (rayIdx >= rayCount || rays[rayIdx].direction == ZERO_VECTOR) return;

	if (!intersections[rayIdx].Hit()) {
		frameBuffer[rayIdx] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
		return;
	}

	Material* materialPtr = &materials[intersections[rayIdx].materialIdx];
	float4 materialColor = materialPtr->color;

	if (materialPtr->type == Material::MaterialType::DIFFUSE) {
		float4 reflection = GetDiffuseReflection(intersections[rayIdx].normal, &rngStates[rayIdx]);
		rays[rayIdx].origin += rays[rayIdx].direction * intersections[rayIdx].t + reflection * EPSILON;
		rays[rayIdx].direction = reflection;

		frameBuffer[rayIdx] *= dot(intersections[rayIdx].normal, reflection) * 2.0f * materialColor;
	}
	else if (materialPtr->type == Material::MaterialType::EMISSIVE) {
		frameBuffer[rayIdx] *= materialColor;

		// Use this as "path finished" until compaction is implemented.
		rays[rayIdx].direction = ZERO_VECTOR;
	}
	else {
		frameBuffer[rayIdx] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);

		// Use this as "path finished" until compaction is implemented.
		rays[rayIdx].direction = ZERO_VECTOR;
	}
}

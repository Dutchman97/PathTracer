#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"

#include <CUDA/hip/hip_vector_types.h>

__global__ void DrawToTexture(hipSurfaceObject_t texture) {
	uint x = threadIdx.x;
	uint y = threadIdx.y;

	// IMPORTANT: Surface functions use bytes for addressing memory; x-coordinate is in bytes.
	// Y-coordinate does not need to be multiplied as the byte offset of the corresponding y-coordinate is internally calculated.
	surf2Dwrite(make_float4(1.0f, 0.2f, 0.2f, 1.0f), texture, x * 4 * 4, y);
}

__global__ void InitializeRng(hiprandStateXORWOW_t* rngStates) {
	uint i = threadIdx.x + blockIdx.x * blockDim.x;

	hiprand_init(1337 + i, 0, 0, &rngStates[i]);
}

__global__ void InitializeRays(Ray* rays, hiprandStateXORWOW_t* rngStates, int screenWidth, int screenHeight, float4 origin, float4 topLeft, float4 topRight, float4 bottomLeft) {
	uint i = threadIdx.x + blockIdx.x * blockDim.x;
	uint x = i % screenWidth;
	uint y = i / screenWidth;

	if (x >= screenWidth || y >= screenHeight) return;

	float xScreen = ((float)x + hiprand_uniform(&rngStates[i])) / screenWidth;
	float yScreen = ((float)y + hiprand_uniform(&rngStates[i])) / screenHeight;

	Ray* rayPtr = &rays[i];
	rayPtr->origin = origin;
	rayPtr->direction = normalize(topLeft + (topRight - topLeft) * xScreen + (bottomLeft - topLeft) * yScreen);
}

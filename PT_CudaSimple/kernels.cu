#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"

#include <CUDA/hip/hip_vector_types.h>

__global__ void DrawToTexture(hipSurfaceObject_t texture) {
	uint x = threadIdx.x;
	uint y = threadIdx.y;

	// IMPORTANT: Surface functions use bytes for addressing memory; x-coordinate is in bytes.
	// Y-coordinate does not need to be multiplied as the byte offset of the corresponding y-coordinate is internally calculated.
	surf2Dwrite(make_float4(1.0f, 0.2f, 0.2f, 1.0f), texture, x * 4 * 4, y);
}

__global__ void Initialize(Ray* rays, size_t rayArrayPitch, int screenWidth, int screenHeight, float4 origin, float4 topLeft, float4 topRight, float4 bottomLeft) {
	uint x = threadIdx.x;
	uint y = threadIdx.y;

	if (x >= screenWidth || y >= screenHeight) return;

	float xScreen = (float)x / screenWidth;
	float yScreen = (float)y / screenHeight;

	Ray* rayPtr = GetFromPitchedMemory(rays, rayArrayPitch, x, y);
	rayPtr->origin = origin;
	rayPtr->direction = topLeft + (topRight - topLeft) * xScreen + (bottomLeft - topLeft) * yScreen;
}
